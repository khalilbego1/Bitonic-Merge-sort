#include "hip/hip_runtime.h"

#include<stdio.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<cmath>
#include<random>
#include<algorithm>
#include"bitonic.hxx"
#include"buildData.h"



template<typename T>
__global__ void bitonicSort(T* data, int N);

int main(int argc, char** argv) {

  if(argc != 4) {
	printf("usage: bitonic <N> <BLOCKS> <THREADS>\n");
	exit(1);
  }

  hipEvent_t start, stop;
  float time_elapsed=0.0;
  int N = pow(2,atoi(argv[1]));
  int BLOCKS = atoi(argv[2]);
  int THREADS = atoi(argv[3]);

  TYPE* h_data = (TYPE*)malloc(N*sizeof(TYPE));

  TYPE* d_data;
  hipMalloc(&d_data, N*sizeof(TYPE));
  float total_time=0.0;

  srand(time(NULL));

  create_random_list<TYPE>(h_data, N, 0);

  hipMemcpy(d_data, h_data, N*sizeof(TYPE), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  bitonicSort<TYPE,cmp>(d_data,N,BLOCKS, THREADS);

  hipDeviceSynchronize();

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_elapsed, start, stop);

  printf("%lf\n", time_elapsed);

  hipMemcpy(h_data, d_data, N*sizeof(TYPE), hipMemcpyDeviceToHost);

#ifdef DEBUG
  bool error=false;
  for(int i=1; i<N; i++) {
    if(h_data[i-1] > h_data[i]) {
      error=true;
      printf("i:%d, %d > %d\n", i,h_data[i-1], h_data[i]);
    }
  }

  if(error)
    printf("NOT SORTED!\n");
  else
    printf("SORTED!\n");
#endif

  hipFree(d_data);
  free(h_data);
}

