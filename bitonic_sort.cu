


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 512 
#define BLOCKS 32768 
#define NUM_VALS THREADS*BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; 
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  if ((ixj)>i) {
    if ((i&k)==0) {
      if (dev_values[i]>dev_values[ixj]) {
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      if (dev_values[i]<dev_values[ixj]) {
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}


void bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS,1);    
  dim3 threads(THREADS,1);  
  int j, k;
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipFree(dev_values);
}

int main(void)
{
  clock_t start, stop;

  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);

  start = clock();
  bitonic_sort(values);
  stop = clock();

  print_elapsed(start, stop);
}
